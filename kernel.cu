#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

struct WavHeader {
    char riff[4];
    uint32_t fileSize;
    char wave[4];
    char fmt[4];
    uint32_t fmtSize;
    uint16_t audioFormat;
    uint16_t numChannels;
    uint32_t sampleRate;
    uint32_t byteRate;
    uint16_t blockAlign;
    uint16_t bitsPerSample;
    char data[4];
    uint32_t dataSize;
};


__global__ void removeNoise(char* data, int dataSize, float threshold, int startSample, int endSample, float noiseReductionFactor = 0.4f) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= startSample && idx < endSample) {
        data[idx] = 0;
    }
    else {
        if (abs(data[idx]) < threshold) {
            float amplitude = static_cast<float>(data[idx]) / 32768.0f;
            amplitude *= noiseReductionFactor;
            data[idx] = static_cast<char>(amplitude * 32768.0f);
        }
    }
}

__global__ void calculateNoiseThresholdKernel(const char* data, int dataSize, int sampleRate, float* result, int startSample, int endSample, uint16_t numChannels) {
    // check 
    startSample = min(startSample, dataSize);
    endSample = min(endSample, dataSize);

    __shared__ float sharedData[1024];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;


    if (i < dataSize) {
        sharedData[tid] = abs(static_cast<float>(data[i]));
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }


    if (tid == 0)
        atomicAdd(result, sharedData[0]);
}

__global__ void increaseVolume(char* data, int dataSize, float gainDb) {
    const float gainLinear = powf(10.0, gainDb / 20.0);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        float sample = static_cast<float>(data[idx]) / 32768.0f;
        sample *= gainLinear;

        if (sample > 1.0f) sample = 1.0f;
        if (sample < -1.0f) sample = -1.0f;

        data[idx] = static_cast<char>(sample * 32768.0f);
    }
}

__global__ void normalizeAudio(char* data, int dataSize, float targetLevel, float threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        float sample = static_cast<float>(data[idx]) / 32768.0f;
        sample *= targetLevel;
        data[idx] = static_cast<char>(sample * 32768.0f);
    }
}

__global__ void addReverb(char* data, int dataSize, int delaySamples, float decay) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int delayedIdx = idx - delaySamples;
    if (idx < dataSize && delayedIdx >= 0) {
        float delayedSample = static_cast<float>(data[delayedIdx]) / 32768.0f;
        float currentSample = static_cast<float>(data[idx]) / 32768.0f;
        float reverbSample = currentSample + delayedSample * decay;
        data[idx] = static_cast<char>(reverbSample * 32768.0f);
    }
}

__global__ void fadeIn(char* data, int dataSize, int fadeLength) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < fadeLength) {
        float fadeFactor = static_cast<float>(idx) / fadeLength;
        float sample = static_cast<float>(data[idx]) / 32768.0f;
        sample *= fadeFactor;
        data[idx] = static_cast<char>(sample * 32768.0f);
    }
}

__global__ void fadeOut(char* data, int dataSize, int fadeLength) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int fadeStartIdx = dataSize - fadeLength;
    if (idx >= fadeStartIdx && idx < dataSize) {
        float fadeFactor = static_cast<float>(dataSize - idx) / fadeLength;
        float sample = static_cast<float>(data[idx]) / 32768.0f;
        sample *= fadeFactor;
        data[idx] = static_cast<char>(sample * 32768.0f);
    }
}

__global__ void changeSpeed(const char* in_data, char* out_data, int in_dataSize, float speedFactor, uint16_t numChannels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int outIdx = idx * numChannels;

    if (outIdx < in_dataSize) {
        float srcIdx = idx * speedFactor;
        int srcIdxInt = static_cast<int>(srcIdx) * numChannels;
        int srcIdxNext = min(srcIdxInt + numChannels, in_dataSize - numChannels);
        float frac = srcIdx - static_cast<int>(srcIdx);

        for (int channel = 0; channel < numChannels; ++channel) {
            if (srcIdxInt + channel < in_dataSize && srcIdxNext + channel < in_dataSize) {
                short sample1 = static_cast<short>((in_data[srcIdxInt + channel] << 8) | (in_data[srcIdxInt + channel + 1] & 0xFF));
                short sample2 = static_cast<short>((in_data[srcIdxNext + channel] << 8) | (in_data[srcIdxNext + channel + 1] & 0xFF));

                float interpSample = (1.0f - frac) * sample1 + frac * sample2;
                interpSample = max(min(interpSample, 32767.0f), -32768.0f);

                out_data[outIdx + channel] = static_cast<char>((static_cast<short>(interpSample) >> 8) & 0xFF);
                out_data[outIdx + channel + 1] = static_cast<char>(static_cast<short>(interpSample) & 0xFF);
            }
        }
    }
}





int main() {
    const char* inputFileName = "input.wav";
    const char* outputFileName = "output.wav";

    // open
    std::ifstream inputFile(inputFileName, std::ios::binary);
    if (!inputFile.is_open()) {
        std::cerr << "Error opening input file." << std::endl;
        return 1;
    }

    // header
    WavHeader header;
    inputFile.read(reinterpret_cast<char*>(&header), sizeof(WavHeader));
    std::cout << "Channels: " << header.numChannels << std::endl;
    std::cout << "Sample Rate: " << header.sampleRate << std::endl;
    std::cout << "Bits Per Sample: " << header.bitsPerSample << std::endl;


    // is supported
    if (header.audioFormat != 1 || header.bitsPerSample != 16) {
        std::cerr << "Unsupported audio format or bits per sample." << std::endl;
        return 1;
    }

    // length
    float audioDuration = static_cast<float>(header.dataSize) / (header.sampleRate * header.numChannels * (header.bitsPerSample / 8));

    std::cout << "Audio Duration: " << audioDuration << " seconds" << std::endl;

    // enter parametrs
    const float targetLevel = 0.85f;
    float startSeconds, endSeconds, gainDb, noiseThreshold, decay, fade, speedFactor;;
    std::cout << "Enter start time in seconds (Noise sample): ";
    std::cin >> startSeconds;
    std::cout << "Enter end time in seconds (Noise sample): ";
    std::cin >> endSeconds;
    std::cout << "Enter gain in dB: ";
    std::cin >> gainDb;
    std::cout << "Enter reverb coeff: ";
    std::cin >> decay;
    std::cout << "Enter fade length: ";
    std::cin >> fade;
    std::cout << "Change speed" << std::endl;
    std::cin >> speedFactor;
    int new_dataSize = static_cast<int>(header.dataSize / speedFactor);
    
    
    // reading data
    std::vector<char> dataBuffer(header.dataSize);
    inputFile.read(dataBuffer.data(), header.dataSize);

    // CUDA malloc
    float* d_result;
    hipMalloc((void**)&d_result, sizeof(float));

    char* d_data;
    hipMalloc((void**)&d_data, header.dataSize);
    hipMemcpy(d_data, dataBuffer.data(), header.dataSize, hipMemcpyHostToDevice);

    char* d_newData;
    hipMalloc((void**)&d_newData, new_dataSize);

    // CUDA grid
    int blockSize;
    int minGridSize, gridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, removeNoise, 0, header.dataSize);
    gridSize = (header.dataSize + blockSize - 1) / blockSize;

    // convert sample2time
    int startSample = static_cast<int>(startSeconds * header.sampleRate * header.numChannels * header.numChannels);
    int endSample = static_cast<int>(endSeconds * header.sampleRate * header.numChannels * header.numChannels);
    int fadeLength = static_cast<int>(fade * header.sampleRate * header.numChannels * header.numChannels);
    int delaySamples = static_cast<int>(0.5f * header.sampleRate);

    // kernels
    calculateNoiseThresholdKernel << <gridSize, blockSize >> > (d_data, header.dataSize, header.sampleRate, d_result, startSample, endSample, header.numChannels);
    hipMemcpy(&noiseThreshold, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Calculate mean over the samples
    int numSamples = endSample - startSample;
    noiseThreshold /= (numSamples * header.numChannels * header.numChannels * header.bitsPerSample);
    std::cout << "Automatic Noise Threshold: " << noiseThreshold << std::endl;

    removeNoise << <gridSize, blockSize >> > (d_data, header.dataSize, noiseThreshold, startSample, endSample);
    hipMemcpy(dataBuffer.data(), d_data, header.dataSize, hipMemcpyDeviceToHost);


    increaseVolume << <gridSize, blockSize >> > (d_data, header.dataSize, gainDb);
    hipMemcpy(dataBuffer.data(), d_data, header.dataSize, hipMemcpyDeviceToHost);


    normalizeAudio << <gridSize, blockSize >> > (d_data, header.dataSize, targetLevel, noiseThreshold);
    hipMemcpy(dataBuffer.data(), d_data, header.dataSize, hipMemcpyDeviceToHost);


    addReverb << <gridSize, blockSize >> > (d_data, header.dataSize, delaySamples, decay);
    hipMemcpy(dataBuffer.data(), d_data, header.dataSize, hipMemcpyDeviceToHost);

    fadeIn << <gridSize, blockSize >> > (d_data, header.dataSize, fadeLength);
    hipMemcpy(dataBuffer.data(), d_data, header.dataSize, hipMemcpyDeviceToHost);
    fadeOut << <gridSize, blockSize >> > (d_data, header.dataSize, fadeLength);
    hipMemcpy(dataBuffer.data(), d_data, header.dataSize, hipMemcpyDeviceToHost);

    gridSize = (new_dataSize + blockSize - 1) / blockSize;
    changeSpeed << <gridSize, blockSize >> > (d_data, d_newData, header.dataSize, speedFactor, header.numChannels);
    header.dataSize = new_dataSize;
    std::vector<char> newDataBuffer(new_dataSize);
    hipMemcpy(newDataBuffer.data(), d_newData, new_dataSize, hipMemcpyDeviceToHost);


    // CUDA free
    hipFree(d_result);
    hipFree(d_data);
    hipFree(d_newData);


    // save output
    std::ofstream outputFile(outputFileName, std::ios::binary);
    if (!outputFile.is_open()) {
        std::cerr << "Error opening output file." << std::endl;
        return 1;
    }

    outputFile.write(reinterpret_cast<const char*>(&header), sizeof(WavHeader));
    outputFile.write(newDataBuffer.data(), new_dataSize);

    std::cout << "File processing completed successfully." << std::endl;

    return 0;
}
